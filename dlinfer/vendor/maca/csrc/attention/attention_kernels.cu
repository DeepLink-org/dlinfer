#include "hip/hip_runtime.h"
// 2024 - Modified by MetaX Integrated Circuits (Shanghai) Co., Ltd. All Rights Reserved.
/*
 * Adapted from
 * https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.hpp
 * Copyright (c) 2023, The vLLM team.
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <algorithm>

#include "attention_dtypes.h"
#include "attention_utils.cuh"

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
  #include "../quantization/fp8/amd/quant_utils.cuh"
typedef __hip_bfloat16 __hip_bfloat16;
#else
  #include "../quantization/fp8/nvidia/quant_utils.cuh"
#endif

#ifndef USE_ROCM
  #define WARP_SIZE 32
#else
  #define WARP_SIZE warpSize
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace vllm {

// Utility function for attention softmax.
template <int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  // Decompose the thread index into warp / lane.
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

  // Compute the sum per warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Warp leaders store the data to shared memory.
  if (lane == 0) {
    red_smem[warp] = sum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The warps compute the final sums.
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }

  // Parallel reduction inside the warp.
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Broadcast to other threads.
  return VLLM_SHFL_SYNC(sum, 0);
}

template<int NUM_WARPS>
inline __device__ float mxblock_sum(float* red_smem, float sum) {
  // Decompose the thread index into warp / lane.
  int warp = threadIdx.x / MXWARP_SIZE;
  int lane = threadIdx.x % MXWARP_SIZE;

  // Compute the sum per warp.
#pragma unroll
  for (int mask = MXWARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += MXVLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Warp leaders store the data to shared memory.
  if (lane == 0) {
    red_smem[warp] = sum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The warps compute the final sums.
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }
 // Parallel reduction inside the warp.
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += MXVLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Broadcast to other threads.
  return MXVLLM_SHFL_SYNC(sum, 0);
}

template<typename scalar_t>
__device__  float __forceinline__ atten_mul(scalar_t *a, float b, int j) {
}

template<>
__device__ float __forceinline__ atten_mul(uint16_t *a, float b, int j) {
    return __half2float(*((half*)a + j)) * __half2float(__float2half(b));
}

template<>
__device__ float __forceinline__ atten_mul(__hip_bfloat16 *a, float b, int j) {
    return __bfloat162float(*(a + j)) * __bfloat162float(__float2bfloat16(b));
}

template<typename scalar_t, typename cache_t>
__device__ float __forceinline__ atten_dot(scalar_t* a, cache_t *b ,int i){

}
template<>
__device__ float __forceinline__ atten_dot(uint16_t* a, uint16_t *b ,int i){
  return __half2float(*((half*)a + i)) * __half2float(*((half*)b + i));
}

template<>
__device__ float __forceinline__ atten_dot(__hip_bfloat16* a, __hip_bfloat16 *b ,int i){
  return __bfloat162float(a[i]) * __bfloat162float(b[i]);
}


// TODO(woosuk): Merge the last two dimensions of the grid.
// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int PARTITION_SIZE = 0>  // Zero means no partitioning.
__device__ void paged_attention_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float k_scale, const float v_scale, const int tp_rank,
    const int blocksparse_local_blocks, const int blocksparse_vert_stride,
    const int blocksparse_block_size, const int blocksparse_head_sliding_step) {
  const int seq_idx = blockIdx.y;
  const int partition_idx = blockIdx.z;
  const int max_num_partitions = gridDim.z;
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int seq_len = seq_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= seq_len) {
    // No work to do. Terminate the thread block.
    return;
  }

  const int num_seq_blocks = DIVIDE_ROUND_UP(seq_len, BLOCK_SIZE);
  const int num_blocks_per_partition =
      USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_seq_blocks;

  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx =
      USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx =
      MIN(start_block_idx + num_blocks_per_partition, num_seq_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx =
      MIN(start_token_idx + num_blocks * BLOCK_SIZE, seq_len);
  const int num_tokens = end_token_idx - start_token_idx;

  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_THREAD_GROUPS =
      NUM_THREADS / THREAD_GROUP_SIZE;  // Note: This assumes THREAD_GROUP_SIZE
                                        // divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP =
      DIVIDE_ROUND_UP(BLOCK_SIZE, WARP_SIZE);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  const float alibi_slope =
      alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];

  constexpr int VEC_SIZE = 16 / sizeof(scalar_t);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Quant_vec = typename Vec<cache_t, VEC_SIZE>::Type;

  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers.
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in
  // the group has 0, 4, 8, ... th vectors of the query, and the second thread
  // has 1, 5, 9, ... th vectors of the query, and so on. NOTE(woosuk): Because
  // q is split from a qkv tensor, it may not be contiguous.
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
#pragma unroll
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD;
       i += NUM_THREAD_GROUPS) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    q_vecs[thread_group_offset][i] =
        *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }
  __syncthreads();  // TODO(naed90): possible speedup if this is replaced with a
                    // memory wall right before we use q_vecs

  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 for the softmax logits for better accuracy.
  float* logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  constexpr int x = 32 / sizeof(cache_t);
  float qk_max = -FLT_MAX;

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;

  // blocksparse specific vars
  int bs_block_offset;
  int q_bs_block_id;
  if constexpr (IS_BLOCK_SPARSE) {
    // const int num_blocksparse_blocks = DIVIDE_ROUND_UP(seq_len,
    // blocksparse_block_size);
    q_bs_block_id = (seq_len - 1) / blocksparse_block_size;
    if (blocksparse_head_sliding_step >= 0)
      // sliding on q heads
      bs_block_offset =
          (tp_rank * num_heads + head_idx) * blocksparse_head_sliding_step + 1;
    else
      // sliding on kv heads
      bs_block_offset = (tp_rank * num_kv_heads + kv_head_idx) *
                            (-blocksparse_head_sliding_step) +
                        1;
  }

  int block_idx0 = start_block_idx + warp_idx;
  int kv_offset0, kv_offset1;
  K_vec load_k[NUM_VECS_PER_THREAD];
  K_vec compute_k[NUM_VECS_PER_THREAD];

  kv_offset0 = block_table[block_idx0];
  if(block_idx0 + NUM_WARPS < end_block_idx) {
    kv_offset1 = block_table[block_idx0 + NUM_WARPS];
  }
  
  for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
    const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
    const int token_idx = block_idx0 * BLOCK_SIZE + physical_block_offset;
    const cache_t* k_ptr = k_cache + static_cast<int64_t>(kv_offset0) * kv_block_stride
                                      + kv_head_idx * kv_head_stride
                                      + physical_block_offset * x;

#pragma unroll
      for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
	const int offset1 = (vec_idx * VEC_SIZE) / x;
        const int offset2 = (vec_idx * VEC_SIZE) % x;
#if 0
        if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          k_vecs[j] = *reinterpret_cast<const K_vec*>(
              k_ptr + offset1 * BLOCK_SIZE * x + offset2);
        } else {
          // Vector conversion from Quant_vec to K_vec.
          Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(
              k_ptr + offset1 * BLOCK_SIZE * x + offset2);
          k_vecs[j] = fp8::scaled_convert<K_vec, Quant_vec, KV_DTYPE>(
              k_vec_quant, k_scale);
        }
#endif
	load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
      }
  }

   for (int block_idx = block_idx0; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    for(int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      #pragma unroll
      for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        compute_k[j] = load_k[j];
      }
      if(block_idx < end_block_idx - NUM_WARPS) {
          kv_offset0 = kv_offset1;
          if(block_idx < end_block_idx - (NUM_WARPS << 1)) {
            kv_offset1 = block_table[block_idx + (NUM_WARPS<<1)];
          }

          const cache_t* k_ptr = k_cache + static_cast<int64_t>(kv_offset0) * kv_block_stride
                                      + kv_head_idx * kv_head_stride
                                      + physical_block_offset * x;
          #pragma unroll NUM_VECS_PER_THREAD
          for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
              const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
              const int offset1 = (vec_idx * VEC_SIZE) / x;
              const int offset2 = (vec_idx * VEC_SIZE) % x;
#if 0        
	      if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
                  k_vecs[j] = *reinterpret_cast<const K_vec*>(
                  k_ptr + offset1 * BLOCK_SIZE * x + offset2);
              } else {
                  // Vector conversion from Quant_vec to K_vec.
                  Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(
                      k_ptr + offset1 * BLOCK_SIZE * x + offset2);
                  k_vecs[j] = fp8::scaled_convert<K_vec, Quant_vec, KV_DTYPE>(
                      k_vec_quant, k_scale);
              }
#endif
	      load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
          }
      }

      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
      // Compute the parallel products for Q*K^T (treat vector lanes separately).
      float qk = 0.0f;
      #pragma unroll
      for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        scalar_t *ptr_q = (scalar_t*)&q_vecs[thread_group_offset][j];
        cache_t *ptr_c = (cache_t*)&compute_k[j];
        #pragma unroll
        for(int k = 0; k < VEC_SIZE; k++) {
          qk += atten_dot(ptr_q,ptr_c,k);
        }
      }

      #pragma unroll
      for (int mask = THREAD_GROUP_SIZE / 2; mask >= 1; mask /= 2) {
        qk += VLLM_SHFL_XOR_SYNC(qk, mask);
      }
      qk = scale * qk;

      // Add the ALiBi bias if slopes are given.
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - seq_len + 1) : 0;

      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        const bool mask = token_idx >= seq_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        // Update the max value.
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }

  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = qk_max;
  }
  __syncthreads();

  // TODO(woosuk): Refactor this part.
  // Get the max qk value for the sequence.
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  // Broadcast the max qk value to all threads.
  qk_max = VLLM_SHFL_SYNC(qk_max, 0);

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);

  // Compute softmax.
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  // If partitioning is enabled, store the max logit and exp_sum.
  if (USE_PARTITIONING && thread_idx == 0) {
    float* max_logits_ptr = max_logits +
                            seq_idx * num_heads * max_num_partitions +
                            head_idx * max_num_partitions + partition_idx;
    *max_logits_ptr = qk_max;
    float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions +
                          head_idx * max_num_partitions + partition_idx;
    *exp_sums_ptr = exp_sum;
  }

  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = 16 / sizeof(scalar_t);
  constexpr int NUM_V_VECS_PER_THREAD = HEAD_SIZE / V_VEC_SIZE;
  constexpr int NUM_COLS_PER_ITER = MAX(WARP_SIZE / NUM_V_VECS_PER_THREAD,1);
  constexpr int NUM_VALID_THREAD = NUM_COLS_PER_ITER * NUM_V_VECS_PER_THREAD;
  constexpr int NUM_LGT_PER_COL = (BLOCK_SIZE + NUM_COLS_PER_ITER - 1) / NUM_COLS_PER_ITER;
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using Float_L_vec = typename FloatVec<L_vec>::Type;
  const int physical_block_offset = lane / NUM_V_VECS_PER_THREAD;
  const int laneid = lane % NUM_V_VECS_PER_THREAD;
  V_vec v_vecs[NUM_LGT_PER_COL];
  V_vec v_prev_vecs[NUM_LGT_PER_COL];
  float accs[V_VEC_SIZE];
  float reg_log[NUM_LGT_PER_COL];
  float reg_prev_log[NUM_LGT_PER_COL];
  #pragma unroll
  for(int i = 0; i < V_VEC_SIZE; i++) {
    accs[i] = 0.0f;
  }
  int token_idx, kv_stride, block_offset;
  kv_stride = BLOCK_SIZE * HEAD_SIZE ;
  kv_offset0 = block_table[block_idx0];
  block_offset = NUM_COLS_PER_ITER * HEAD_SIZE;
  if(block_idx0 + NUM_WARPS < end_block_idx) {
    kv_offset1 = block_table[block_idx0 + NUM_WARPS];
  }
  token_idx = block_idx0 * BLOCK_SIZE + physical_block_offset;
  const cache_t *v_ptr0 = v_cache + kv_head_idx * kv_stride + physical_block_offset * HEAD_SIZE; 
  const cache_t* v_ptr = v_ptr0 + static_cast<int64_t>(kv_offset0) * kv_block_stride;
  float *ptr_logits = logits + token_idx - start_token_idx;
  if(lane < NUM_VALID_THREAD) {
    if(block_idx0 == num_seq_blocks - 1) {
    #pragma unroll
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
        if(token_idx + i * NUM_COLS_PER_ITER < seq_len ) {
          const int idx = laneid * V_VEC_SIZE + i * block_offset;
          v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
          reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
        }
      }
    } else {
      #pragma unroll
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
        if(token_idx + i * NUM_COLS_PER_ITER < seq_len ) {
          const int idx = laneid * V_VEC_SIZE + i * block_offset;
          v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
          reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
        }
      }
    }


  for(int block_idx = block_idx0; block_idx < end_block_idx; block_idx += NUM_WARPS) {
      int next_block = block_idx + NUM_WARPS;
      int nnext_block = next_block + NUM_WARPS;
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
          v_vecs[i] = v_prev_vecs[i];
          reg_log[i] = reg_prev_log[i];
      }
      if(next_block < end_block_idx) {
          kv_offset0 = kv_offset1;
          if(nnext_block < end_block_idx) {
              kv_offset1 = block_table[nnext_block];
          }
          token_idx = next_block * BLOCK_SIZE + physical_block_offset;
          const cache_t* v_ptr = v_ptr0 + static_cast<int64_t>(kv_offset0) * kv_block_stride;
          ptr_logits = logits + token_idx - start_token_idx;
          if(next_block == num_seq_blocks - 1) {
              #pragma unroll
              for(int i = 0; i < NUM_LGT_PER_COL; i++) {
                  if(token_idx + i * NUM_COLS_PER_ITER < seq_len && i * NUM_COLS_PER_ITER + physical_block_offset < BLOCK_SIZE) {
                      const int idx = laneid * V_VEC_SIZE + i * block_offset;
#if 0
  		      if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          	          v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
                      } else {
                          V_quant_vec v_quant_vec =
                              *reinterpret_cast<const V_quant_vec*>(v_ptr + offset);
                          // Vector conversion from V_quant_vec to V_vec.
                          v_vec = fp8::scaled_convert<V_vec, V_quant_vec, KV_DTYPE>(v_quant_vec,
                                                                    v_scale);
        	      }
#endif
	              v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
                      reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
                  }
              }
          } else {
              #pragma unroll
              for(int i = 0; i < NUM_LGT_PER_COL; i++) {
                  if(token_idx + i * NUM_COLS_PER_ITER < seq_len && i * NUM_COLS_PER_ITER + physical_block_offset < BLOCK_SIZE) {
                      const int idx = laneid * V_VEC_SIZE + i * block_offset;
#if 0
  		      if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          	          v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
                      } else {
                          V_quant_vec v_quant_vec =
                              *reinterpret_cast<const V_quant_vec*>(v_ptr + offset);
                          // Vector conversion from V_quant_vec to V_vec.
                          v_vec = fp8::scaled_convert<V_vec, V_quant_vec, KV_DTYPE>(v_quant_vec,
                                                                    v_scale);
        	      }
#endif
	              v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
                      reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
                  }
              }
          }
      }
      token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
          if(token_idx + i * NUM_COLS_PER_ITER < seq_len && i * NUM_COLS_PER_ITER + physical_block_offset < BLOCK_SIZE) {
              scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vecs[i]);
              for(int j = 0; j < V_VEC_SIZE; j++) {
                  accs[j] += atten_mul(v_vec_ptr, reg_log[i], j);
	      }
           }
        }
     }
  }
  __syncthreads();
  //need move
  float* out_smem = reinterpret_cast<float*>(shared_mem);
  for(int i = threadIdx.x; i < NUM_WARPS * NUM_COLS_PER_ITER * HEAD_SIZE; i += blockDim.x) {
    out_smem[i] = 0.0f;
  }
  __syncthreads(); 

  if(lane < NUM_VALID_THREAD) {
    float*ptr_out_smem = out_smem + warp_idx * HEAD_SIZE*NUM_COLS_PER_ITER + physical_block_offset * HEAD_SIZE + laneid* V_VEC_SIZE;
    for(int i = 0; i < V_VEC_SIZE; i++) {
      ptr_out_smem[i] = accs[i];
    }
  }
   __syncthreads();
  for(int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
    float r = 0;
    #pragma unroll
    for(int j = 0; j < NUM_WARPS * NUM_COLS_PER_ITER; j++){
        r += out_smem[j * HEAD_SIZE + i];
    }
    scalar_t* out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
                        + head_idx * max_num_partitions * HEAD_SIZE
                        + partition_idx * HEAD_SIZE;
    from_float(*(out_ptr + i), r);
  }

}
/*
  // NOTE(woosuk): A barrier is required because the shared memory space for
  // logits is reused for the output.
  __syncthreads();

  // Perform reduction across warps.
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    // Upper warps write to shared memory.
    if (warp_idx >= mid && warp_idx < i) {
      float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[i];
        }
      }
    }
    __syncthreads();

    // Lower warps update the output.
    if (warp_idx < mid) {
      const float* src = &out_smem[warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }

  // Write the final output.
  if (warp_idx == 0) {
    scalar_t* out_ptr =
        out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE + partition_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        from_float(*(out_ptr + row_idx), accs[i]);
      }
    }
  }
}
*/

template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int PARTITION_SIZE = 0>  // Zero means no partitioning.
__device__ void paged_attention_kernel_32N(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads, head_size/x, block_size, x]->[num_blocks, num_kv_heads, head_size/16, block_size, 16]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads, head_size, block_size]->[num_blocks, num_kv_heads, block_size, head_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  const int seq_idx = blockIdx.y;
  const int partition_idx = blockIdx.z;
  const int max_num_partitions = gridDim.z;
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int seq_len = seq_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= seq_len) {
    // No work to do. Terminate the thread block.
    return;
  }
  const int num_seq_blocks = DIVIDE_ROUND_UP(seq_len, BLOCK_SIZE);
  const int num_blocks_per_partition =
      USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_seq_blocks;

  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx =
      USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx =
      MIN(start_block_idx + num_blocks_per_partition, num_seq_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx =
      MIN(start_token_idx + num_blocks * BLOCK_SIZE, seq_len);
  const int num_tokens = end_token_idx - start_token_idx;
  constexpr int THREAD_GROUP_SIZE = MAX(MXWARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_THREAD_GROUPS =
      NUM_THREADS / THREAD_GROUP_SIZE;  // Note: This assumes THREAD_GROUP_SIZE
                                        // divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP =
	      DIVIDE_ROUND_UP(BLOCK_SIZE, MXWARP_SIZE);
  constexpr int NUM_WARPS = NUM_THREADS / MXWARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / MXWARP_SIZE;
  const int lane = thread_idx % MXWARP_SIZE;

  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  const float alibi_slope =
      alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];

  constexpr int VEC_SIZE = 16 / sizeof(scalar_t);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers.
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in
  // the group has 0, 4, 8, ... th vectors of the query, and the second thread
  // has 1, 5, 9, ... th vectors of the query, and so on. NOTE(woosuk): Because
  // q is split from a qkv tensor, it may not be contiguous.
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
#pragma unroll
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD;
       i += NUM_THREAD_GROUPS) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    q_vecs[thread_group_offset][i] =
        *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }
  __syncthreads();  // TODO(naed90): possible speedup if this is replaced with a
                    // memory wall right before we use q_vecs
  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 for the softmax logits for better accuracy.
  float* logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  constexpr int x = 32 / sizeof(cache_t);      // VLLM_0.4.0  x=32
  float qk_max = -FLT_MAX;

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
#if 0
  // blocksparse specific vars
  int bs_block_offset;
  int q_bs_block_id;
  if constexpr (IS_BLOCK_SPARSE) {
    // const int num_blocksparse_blocks = DIVIDE_ROUND_UP(seq_len,
    // blocksparse_block_size);
    q_bs_block_id = (seq_len - 1) / blocksparse_block_size;
    if (blocksparse_head_sliding_step >= 0)
      // sliding on q heads
      bs_block_offset =
          (tp_rank * num_heads + head_idx) * blocksparse_head_sliding_step + 1;
    else
      // sliding on kv heads
      bs_block_offset = (tp_rank * num_kv_heads + kv_head_idx) *
                            (-blocksparse_head_sliding_step) +
                        1;
  }
#endif
  int block_idx0 = start_block_idx + warp_idx;
  int kv_offset0, kv_offset1;
  K_vec load_k[NUM_VECS_PER_THREAD];
  K_vec compute_k[NUM_VECS_PER_THREAD];
  kv_offset0 = block_table[block_idx0];
  if(block_idx0 + NUM_WARPS < end_block_idx) {
    kv_offset1 = block_table[block_idx0 + NUM_WARPS];
  }
  
  for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
    const int physical_block_offset = (thread_group_idx + i * MXWARP_SIZE) % BLOCK_SIZE;
    const int token_idx = block_idx0 * BLOCK_SIZE + physical_block_offset;
    const cache_t* k_ptr = k_cache + static_cast<int64_t>(kv_offset0) * kv_block_stride
                                      + kv_head_idx * kv_head_stride
                                      + physical_block_offset * x;

#pragma unroll
    for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
      const int vec_idx = (thread_group_offset + j * THREAD_GROUP_SIZE) * VEC_SIZE;
      const int offset1 = vec_idx / x;
      const int offset2 = vec_idx % x;
#if 0
      if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          // Vector conversion from Quant_vec to K_vec.
          Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(
              k_ptr + offset1 * BLOCK_SIZE * x + offset2);
	  load_k[j] = fp8_e5m2_unscaled::vec_conversion<K_vec, Quant_vec>(k_vec_quant);
        } else {
	  load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
        }
#endif
      load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
    }
  }

   for (int block_idx = block_idx0; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    for(int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset = (thread_group_idx + i * MXWARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      #pragma unroll
      for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        compute_k[j] = load_k[j];
      }
      if(block_idx < end_block_idx - NUM_WARPS) {
          kv_offset0 = kv_offset1;
          if(block_idx < end_block_idx - (NUM_WARPS << 1)) {
            kv_offset1 = block_table[block_idx + (NUM_WARPS<<1)];
          }

          const cache_t* k_ptr = k_cache + static_cast<int64_t>(kv_offset0) * kv_block_stride
                                      + kv_head_idx * kv_head_stride
                                      + physical_block_offset * x;
          #pragma unroll NUM_VECS_PER_THREAD
          for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
	      const int vec_idx = (thread_group_offset + j * THREAD_GROUP_SIZE) * VEC_SIZE;
              const int offset1 = vec_idx / x;
              const int offset2 = vec_idx % x;
#if 0
              if constexpr (IS_FP8_E5M2_KV_CACHE) {
      #ifdef ENABLE_FP8_E5M2
                Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
                // Vector conversion from Quant_vec to K_vec.
                load_k[j] = fp8_e5m2_unscaled::vec_conversion<K_vec, Quant_vec>(k_vec_quant);
      #else
                assert(false);
      #endif
              } else {
                load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
              }
#endif
                load_k[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
          }
      }
      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
      // Compute the parallel products for Q*K^T (treat vector lanes separately).
      float qk = 0.0f;
      #pragma unroll
      for(int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        scalar_t *ptr_q = (scalar_t*)&q_vecs[thread_group_offset][j];
        cache_t *ptr_c = (cache_t*)&compute_k[j];
        #pragma unroll
        for(int k = 0; k < VEC_SIZE; k++) {
          qk += atten_dot(ptr_q,ptr_c,k);
        }
      }
  
      #pragma unroll
      for (int mask = THREAD_GROUP_SIZE / 2; mask >= 1; mask /= 2) {
	qk += MXVLLM_SHFL_XOR_SYNC(qk, mask);
      }
      qk = scale * qk;
      // Add the ALiBi bias if slopes are given.
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - seq_len + 1) : 0;

      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        const bool mask = token_idx >= seq_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        // Update the max value.
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }
  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
#pragma unroll
  for (int mask = MXWARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, MXVLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  if (lane == 0) {
   red_smem[warp_idx] = qk_max;
  }
  __syncthreads();

  // TODO(woosuk): Refactor this part.
  // Get the max qk value for the sequence.
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, MXVLLM_SHFL_XOR_SYNC(qk_max, mask));  // XuBW
  }
  // Broadcast the max qk value to all threads.
  qk_max = MXVLLM_SHFL_SYNC(qk_max, 0);  // XuBW

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = mxblock_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);  // XuBW

  // Compute softmax.
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  // If partitioning is enabled, store the max logit and exp_sum.
  if (USE_PARTITIONING && thread_idx == 0) {
    float* max_logits_ptr = max_logits +
                            seq_idx * num_heads * max_num_partitions +
                            head_idx * max_num_partitions + partition_idx;
    *max_logits_ptr = qk_max;
    float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions +
                          head_idx * max_num_partitions + partition_idx;
    *exp_sums_ptr = exp_sum;
  }
  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = 16 / sizeof(scalar_t);
  constexpr int NUM_V_VECS_PER_THREAD = HEAD_SIZE / V_VEC_SIZE;
  constexpr int NUM_COLS_PER_ITER = MAX(MXWARP_SIZE / NUM_V_VECS_PER_THREAD,1);  // XuBW
  //constexpr int NUM_VALID_THREAD = NUM_COLS_PER_ITER * NUM_V_VECS_PER_THREAD;
  constexpr int NUM_LGT_PER_COL = BLOCK_SIZE / NUM_COLS_PER_ITER;
  //constexpr int NUM_LGT_PER_COL = (BLOCK_SIZE + NUM_COLS_PER_ITER - 1) / NUM_COLS_PER_ITER;
  constexpr int NUM_LANE = NUM_WARPS * NUM_COLS_PER_ITER; // XuBW
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  //using Float_L_vec = typename FloatVec<L_vec>::Type;
  const int physical_block_offset = lane / NUM_V_VECS_PER_THREAD;
  const int laneid = lane % NUM_V_VECS_PER_THREAD;
  V_vec v_vecs[NUM_LGT_PER_COL];
  V_vec v_prev_vecs[NUM_LGT_PER_COL];
  float accs[V_VEC_SIZE];
  float reg_log[NUM_LGT_PER_COL];
  float reg_prev_log[NUM_LGT_PER_COL];
  #pragma unroll
  for(int i = 0; i < V_VEC_SIZE; i++) {
    accs[i] = 0.0f;
  }
  int token_idx, kv_stride, block_offset;
  kv_stride = BLOCK_SIZE * HEAD_SIZE ;
  kv_offset0 = block_table[block_idx0];
  block_offset = NUM_COLS_PER_ITER * HEAD_SIZE;
  if(block_idx0 + NUM_WARPS < end_block_idx) {
    kv_offset1 = block_table[block_idx0 + NUM_WARPS];
  }
  token_idx = block_idx0 * BLOCK_SIZE + physical_block_offset;
  const cache_t *v_ptr0 = v_cache + kv_head_idx * kv_stride + physical_block_offset * HEAD_SIZE; 
  const cache_t* v_ptr = v_ptr0 + static_cast<int64_t>(kv_offset0) * kv_block_stride;
  float *ptr_logits = logits + token_idx - start_token_idx;
  #pragma unroll
  for(int i = 0; i < NUM_LGT_PER_COL; i++) {
     // XuBW
     if(token_idx + i * NUM_COLS_PER_ITER < seq_len ) {
       const int idx = laneid * V_VEC_SIZE + i * block_offset;
       v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
       reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
     }
  } 

  for(int block_idx = block_idx0; block_idx < end_block_idx; block_idx += NUM_WARPS) {
      int next_block = block_idx + NUM_WARPS;
      int nnext_block = next_block + NUM_WARPS;
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
          v_vecs[i] = v_prev_vecs[i];
          reg_log[i] = reg_prev_log[i];
      }
      if(next_block < end_block_idx) {
        kv_offset0 = kv_offset1;
        if(nnext_block < end_block_idx) {
          kv_offset1 = block_table[nnext_block];
        }
        token_idx = next_block * BLOCK_SIZE + physical_block_offset;
        const cache_t* v_ptr = v_ptr0 + static_cast<int64_t>(kv_offset0) * kv_block_stride;
        ptr_logits = logits + token_idx - start_token_idx;
        if(next_block == num_seq_blocks - 1) {
          #pragma unroll
          for(int i = 0; i < NUM_LGT_PER_COL; i++) {
            if(token_idx + i * NUM_COLS_PER_ITER < seq_len) {
              const int idx = laneid * V_VEC_SIZE + i * block_offset;
#if 0
              if constexpr (IS_FP8_E5M2_KV_CACHE) {
    #ifdef ENABLE_FP8_E5M2
            V_quant_vec v_quant_vec = *reinterpret_cast<const V_quant_vec*>(v_ptr + idx);
            // Vector conversion from V_quant_vec to V_vec.
            v_prev_vecs[i] = fp8_e5m2_unscaled::vec_conversion<V_vec, V_quant_vec>(v_quant_vec);
    #else
            assert(false);
    #endif
              } else {
                  v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
              }
#endif
	      v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
              reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
            }
          }
        } else {
          #pragma unroll
          for(int i = 0; i < NUM_LGT_PER_COL; i++) {
              const int idx = laneid * V_VEC_SIZE + i * block_offset;
#if 0
              if constexpr (IS_FP8_E5M2_KV_CACHE) {
      #ifdef ENABLE_FP8_E5M2
              V_quant_vec v_quant_vec = *reinterpret_cast<const V_quant_vec*>(v_ptr + idx);
              // Vector conversion from V_quant_vec to V_vec.
              v_prev_vecs[i] = fp8_e5m2_unscaled::vec_conversion<V_vec, V_quant_vec>(v_quant_vec);
      #else
              assert(false);
      #endif
            } else {
              v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
            }
#endif
	      v_prev_vecs[i] = *reinterpret_cast<const V_vec*>(v_ptr + idx);
              reg_prev_log[i] = ptr_logits[i * NUM_COLS_PER_ITER];
            }
          }
        }

      token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      float *ptr_logits = logits + token_idx - start_token_idx;
      for(int i = 0; i < NUM_LGT_PER_COL; i++) {
        if(token_idx + i * NUM_COLS_PER_ITER < seq_len) {
          scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vecs[i]);
          for(int j = 0; j < V_VEC_SIZE; j++) {
            accs[j] += atten_mul(v_vec_ptr, reg_log[i], j);
          }
        }
      }
    }

  __syncthreads();
  //need move
  float* out_smem = reinterpret_cast<float*>(shared_mem);
  float*ptr_out_smem = out_smem + warp_idx * HEAD_SIZE*NUM_COLS_PER_ITER + physical_block_offset * HEAD_SIZE + laneid* V_VEC_SIZE;
  for(int i = 0; i < V_VEC_SIZE; i++) {
      ptr_out_smem[i] = accs[i];
  }
  __syncthreads();
 scalar_t* out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
                        + head_idx * max_num_partitions * HEAD_SIZE
                        + partition_idx * HEAD_SIZE;

  for(int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
    float r = 0;
    #pragma unroll
    for(int j = 0; j < NUM_LANE; j++){
        r += out_smem[j * HEAD_SIZE + i];
    }
    from_float(*(out_ptr + i), r);
  }
}





// Grid: (num_heads, num_seqs, 1).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE>
__global__ void paged_attention_v1_kernel(
    scalar_t* __restrict__ out,           // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float k_scale, const float v_scale, const int tp_rank,
    const int blocksparse_local_blocks, const int blocksparse_vert_stride,
    const int blocksparse_block_size, const int blocksparse_head_sliding_step) {
  paged_attention_kernel<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE>(
      /* exp_sums */ nullptr, /* max_logits */ nullptr, out, q, k_cache,
      v_cache, num_kv_heads, scale, block_tables, seq_lens,
      max_num_blocks_per_seq, alibi_slopes, q_stride, kv_block_stride,
      kv_head_stride, k_scale, v_scale, tp_rank, blocksparse_local_blocks,
      blocksparse_vert_stride, blocksparse_block_size,
      blocksparse_head_sliding_step);
}

template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE>
__global__ void paged_attention_v1_32N_kernel(
    scalar_t* __restrict__ out,           // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  paged_attention_kernel_32N<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE>(
      /* exp_sums */ nullptr, /* max_logits */ nullptr, out, q, k_cache,
      v_cache, num_kv_heads, scale, block_tables, seq_lens,
      max_num_blocks_per_seq, alibi_slopes, q_stride, kv_block_stride,
      kv_head_stride, kv_scale, tp_rank, blocksparse_local_blocks,
      blocksparse_vert_stride, blocksparse_block_size,
      blocksparse_head_sliding_step);
}


// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int PARTITION_SIZE>
__global__ void paged_attention_v2_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,       // [num_seqs, num_heads,
                                          // max_num_partitions]
    scalar_t* __restrict__ tmp_out,       // [num_seqs, num_heads,
                                          // max_num_partitions, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float k_scale, const float v_scale, const int tp_rank,
    const int blocksparse_local_blocks, const int blocksparse_vert_stride,
    const int blocksparse_block_size, const int blocksparse_head_sliding_step) {
  paged_attention_kernel<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE, PARTITION_SIZE>(
      exp_sums, max_logits, tmp_out, q, k_cache, v_cache, num_kv_heads, scale,
      block_tables, seq_lens, max_num_blocks_per_seq, alibi_slopes, q_stride,
      kv_block_stride, kv_head_stride, k_scale, v_scale, tp_rank,
      blocksparse_local_blocks, blocksparse_vert_stride, blocksparse_block_size,
      blocksparse_head_sliding_step);
}

template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int PARTITION_SIZE>
__global__ void paged_attention_v2_32N_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,       // [num_seqs, num_heads,
                                          // max_num_partitions]
    scalar_t* __restrict__ tmp_out,       // [num_seqs, num_heads,
                                          // max_num_partitions, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  paged_attention_kernel_32N<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE, PARTITION_SIZE>(
      exp_sums, max_logits, tmp_out, q, k_cache, v_cache, num_kv_heads, scale,
      block_tables, seq_lens, max_num_blocks_per_seq, alibi_slopes, q_stride,
      kv_block_stride, kv_head_stride, kv_scale, tp_rank,
      blocksparse_local_blocks, blocksparse_vert_stride, blocksparse_block_size,
      blocksparse_head_sliding_step);
}


// Grid: (num_heads, num_seqs).
template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE>
__global__ void paged_attention_v2_reduce_kernel(
    scalar_t* __restrict__ out,            // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads,
                                           // max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                           // max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads,
                                           // max_num_partitions, head_size]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int seq_len = seq_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(seq_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    // No need to reduce. Only copy tmp_out to out.
    scalar_t* out_ptr =
        out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    const scalar_t* tmp_out_ptr =
        tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE;
    for (int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
      out_ptr[i] = tmp_out_ptr[i];
    }
    // Terminate the thread block.
    return;
  }

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warp_idx = threadIdx.x / WARP_SIZE;
  const int lane = threadIdx.x % WARP_SIZE;

  // Size: 2 * num_partitions.
  extern __shared__ char shared_mem[];
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // Load max logits to shared memory.
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  const float* max_logits_ptr = max_logits +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions;
  float max_logit = -FLT_MAX;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    const float l = max_logits_ptr[i];
    shared_max_logits[i] = l;
    max_logit = fmaxf(max_logit, l);
  }
  __syncthreads();

  // Get the global max logit.
  // Reduce within the warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = max_logit;
  }
  __syncthreads();
  // Reduce across warps.
  max_logit = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  // Broadcast the max value to all threads.
  max_logit = VLLM_SHFL_SYNC(max_logit, 0);

  // Load rescaled exp sums to shared memory.
  float* shared_exp_sums =
      reinterpret_cast<float*>(shared_mem + sizeof(float) * num_partitions);
  const float* exp_sums_ptr = exp_sums +
                              seq_idx * num_heads * max_num_partitions +
                              head_idx * max_num_partitions;
  float global_exp_sum = 0.0f;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    float l = shared_max_logits[i];
    float rescaled_exp_sum = exp_sums_ptr[i] * expf(l - max_logit);
    global_exp_sum += rescaled_exp_sum;
    shared_exp_sums[i] = rescaled_exp_sum;
  }
  __syncthreads();
  global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.0f, global_exp_sum + 1e-6f);

  // Aggregate tmp_out to out.
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
      head_idx * max_num_partitions * HEAD_SIZE;
  scalar_t* out_ptr =
      out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
#pragma unroll
  for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
    float acc = 0.0f;
    for (int j = 0; j < num_partitions; ++j) {
      acc += to_float(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] *
             inv_global_exp_sum;
    }
    from_float(out_ptr[i], acc);
  }
}

}  // namespace vllm

#define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, IS_BLOCK_SPARSE>),  \
      shared_mem_size);                                                     \
  vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE>   \
      <<<grid, block, shared_mem_size, stream>>>(                           \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          k_scale, v_scale, tp_rank, blocksparse_local_blocks,              \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step);

#define LAUNCH_PAGED_ATTENTION_V1_32N(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_32N_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, IS_BLOCK_SPARSE>),  \
      shared_mem_size);                                                     \
  vllm::paged_attention_v1_32N_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE>   \
      <<<grid, block, shared_mem_size, stream>>>(                           \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          k_scale, tp_rank, blocksparse_local_blocks,                      \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step);


template< typename scalar_t>
__global__ void reshape_k_layout_new(scalar_t * __restrict__ k_buffer, scalar_t* k_output,int num_blocks,int num_kv_heads, int head_size,int block_size, int x,int dst_x) {
  int k_head_stride = head_size * block_size;
  scalar_t *ptr_k_buffer = k_buffer + blockIdx.x * k_head_stride;
  scalar_t *ptr_output = k_output + blockIdx.x * k_head_stride;
  for(int t = threadIdx.x; t < k_head_stride; t += blockDim.x) {
    int heightId = t / (block_size * dst_x);
    int remain = t % (block_size * dst_x);
    int blockId = remain / dst_x;
    int wId = remain % dst_x;
    int inId = heightId * dst_x + wId;
    int in_y = inId / x;
    int in_x = inId % x;
    int inIndex = in_y  * block_size * x + blockId * x + in_x;
    ptr_output[t] = ptr_k_buffer[inIndex];
  }
}

// [num_blocks, num_kv_heads, head_size, block_size] -->   [num_blocks,  num_kv_heads, block_size,head_size]
template<typename scalar_t>
__global__ void reshape_v_layout(scalar_t * __restrict__ v_buffer, scalar_t* v_output,int num_blocks,int num_kv_heads, int head_size,int block_size) {
      int v_block_stride = head_size * block_size * num_kv_heads;
      int v_head_stride = head_size * block_size;
      scalar_t *ptr_in = v_buffer + blockIdx.x * v_block_stride;
      scalar_t *ptr_output = v_output + blockIdx.x * v_block_stride;
      for(int t = threadIdx.x; t < v_block_stride; t += blockDim.x) {
        int num_kv_headIdx = t / v_head_stride;
        int remain = t % v_head_stride;
        int headId_H = remain / block_size;
        remain = remain % block_size;
        int out_idx = num_kv_headIdx * head_size * block_size + remain * head_size + headId_H;
        ptr_output[out_idx] = ptr_in[t];
      }
}

template<
  typename CACHE_T,
  int BLOCK_SIZE>
void reshape_kv_cache(
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  torch::Tensor& key_cache_new_layer,
  torch::Tensor& value_cache_new_layer,
  int num_seqs,
  int num_heads,
  int head_size,
  int num_kv_heads) {
  int kv_block_stride = key_cache.stride(0); // NU ,BLC ,HEAD, HEAD_DIM
  int kv_head_stride = key_cache.stride(1);

  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  CACHE_T* key_cache_tmp = reinterpret_cast<CACHE_T*>(key_cache_new_layer.data_ptr());
  CACHE_T* value_cache_tmp = reinterpret_cast<CACHE_T*>(value_cache_new_layer.data_ptr());

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  reshape_k_layout_new<CACHE_T><<<dim3(key_cache.size(0)*num_kv_heads,1,1),dim3(256,1,1),0,stream>>>(key_cache_ptr,key_cache_tmp,key_cache.size(0),num_kv_heads,head_size,BLOCK_SIZE,8,16);
  reshape_v_layout<CACHE_T><<<dim3(key_cache.size(0),1,1),dim3(256,1,1),0,stream>>>(value_cache_ptr,value_cache_tmp,key_cache.size(0),num_kv_heads,head_size,BLOCK_SIZE);
}

#define CALL_RESHAPE_LAUNCHER(CACHE_T, BLOCK_SIZE)       \
  reshape_kv_cache<CACHE_T, BLOCK_SIZE>( \
    key_cache,                                                               \
    value_cache,                                                             \
    key_cache_new_layer,                                                     \
    value_cache_new_layer,                                                   \
    num_seqs,\
    num_heads,\
    head_size,\
    num_kv_heads);

#define CALL_RESHAPE_BLOCK_SIZE(CACHE_T) \
  switch (block_size) {                                               \
    case 8:                                                           \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 8);          \
      break;                                                          \
    case 16:                                                          \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 16);         \
      break;                                                          \
    case 32:                                                          \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 32);         \
      break;                                                          \
    default:                                                          \
      TORCH_CHECK(false, "Unsupported block size: ", block_size);     \
      break;                                                          \
  }

void page_reshape_kv_cache(
  torch::Tensor& key_cache,       // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& key_cache_new_layer, //[num_blocks, num_heads, head_size/16, block_size, 16]
  torch::Tensor& value_cache_new_layer,//[num_blocks, num_heads, block_size, head_size]
  // XuBW int -> int64_t
  int num_seqs,
  int num_heads,
  int head_size,
  int num_kv_heads,               // [num_heads]
  int block_size,
  const std::string& kv_cache_dtype) {
  if (kv_cache_dtype == "auto") {
    if (sizeof(key_cache.dtype())==4) {
      //CALL_RESHAPE_BLOCK_SIZE(float);
    } else if (sizeof(key_cache.dtype()) == 2) {
      //CALL_RESHAPE_BLOCK_SIZE(uint16_t);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", key_cache.dtype());
    }
  }  else {
    TORCH_CHECK(false, "Unsupported data type of kv cache: ", kv_cache_dtype);
  }
}



// TODO(woosuk): Tune NUM_THREADS.
template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE,
          int NUM_THREADS = 128>
void paged_attention_v1_launcher(
    torch::Tensor& out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, float k_scale,
    float v_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);  // num head head_dim
  int kv_block_stride = key_cache.stride(0); // NU ,BLC ,HEAD, HEAD_DIM
  int kv_head_stride = key_cache.stride(1);

  [[maybe_unused]] int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_seq_len =
      DIVIDE_ROUND_UP(max_seq_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_seq_len * sizeof(float);
  int V_VEC_SIZE = 16 / sizeof(CACHE_T);
  int NUM_V_VECS_PER_THREAD = head_size / V_VEC_SIZE;
  int NUM_COLS_PER_ITER = MAX(WARP_SIZE / NUM_V_VECS_PER_THREAD, 1);
  int outputs_size = NUM_WARPS * head_size * sizeof(float) * NUM_COLS_PER_ITER;

  // Python-side check in vllm.worker.worker._check_if_can_support_max_seq_len
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_seqs, 1);
  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model. However, we can easily extend this
    // to support any head size which is a multiple of 16.
    case 64:
      LAUNCH_PAGED_ATTENTION_V1_32N(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V1(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V1(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V1(112);
      break;
    case 120:
      LAUNCH_PAGED_ATTENTION_V1(120);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V1_32N(128);
      break;
    case 192:
      LAUNCH_PAGED_ATTENTION_V1(192);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V1_32N(256);
      break;
    case 576:
      LAUNCH_PAGED_ATTENTION_V1_32N(576);
      break;
    default:
      TORCH_CHECK(false, "Unsupported head size: ", head_size);
      break;
  }
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)  \
  paged_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,              \
                              IS_BLOCK_SPARSE>(                              \
      out, query, key_cache, value_cache, num_kv_heads, scale, block_tables, \
      seq_lens, max_seq_len, alibi_slopes, k_scale, v_scale, tp_rank,        \
      blocksparse_local_blocks, blocksparse_vert_stride,                     \
      blocksparse_block_size, blocksparse_head_sliding_step);

#define CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  switch (is_block_sparse) {                                               \
    case true:                                                             \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);     \
      break;                                                               \
    case false:                                                            \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);    \
      break;                                                               \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 8:                                                       \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 8, KV_DTYPE);         \
      break;                                                      \
    case 16:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    case 32:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 32, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

void paged_attention_v1(
    torch::Tensor& out,    // [num_seqs, num_heads, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double k_scale, double v_scale,
    const int64_t tp_rank, const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);

  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V1_LAUNCHER_BLOCK_SIZE)
}

#define LAUNCH_PAGED_ATTENTION_V2(HEAD_SIZE)                                   \
  vllm::paged_attention_v2_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,           \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE,      \
                                  PARTITION_SIZE>                              \
      <<<grid, block, shared_mem_size, stream>>>(                              \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, query_ptr, key_cache_ptr, \
          value_cache_ptr, num_kv_heads, scale, block_tables_ptr,              \
          seq_lens_ptr, max_num_blocks_per_seq, alibi_slopes_ptr, q_stride,    \
          kv_block_stride, kv_head_stride, k_scale, v_scale, tp_rank,          \
          blocksparse_local_blocks, blocksparse_vert_stride,                   \
          blocksparse_block_size, blocksparse_head_sliding_step);              \
  vllm::paged_attention_v2_reduce_kernel<T, HEAD_SIZE, NUM_THREADS,            \
                                         PARTITION_SIZE>                       \
      <<<reduce_grid, block, reduce_shared_mem_size, stream>>>(                \
          out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr, seq_lens_ptr,    \
          max_num_partitions);

#define LAUNCH_PAGED_ATTENTION_V2_32N(HEAD_SIZE)                                   \
  vllm::paged_attention_v2_32N_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,           \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE,      \
                                  PARTITION_SIZE>                              \
      <<<grid, block, shared_mem_size, stream>>>(                              \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, query_ptr, key_cache_ptr, \
          value_cache_ptr, num_kv_heads, scale, block_tables_ptr,              \
          seq_lens_ptr, max_num_blocks_per_seq, alibi_slopes_ptr, q_stride,    \
          kv_block_stride, kv_head_stride, k_scale, tp_rank,                  \
          blocksparse_local_blocks, blocksparse_vert_stride,                   \
          blocksparse_block_size, blocksparse_head_sliding_step);              \
  vllm::paged_attention_v2_reduce_kernel<T, HEAD_SIZE, NUM_THREADS,            \
                                         PARTITION_SIZE>                       \
      <<<reduce_grid, block, reduce_shared_mem_size, stream>>>(                \
          out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr, seq_lens_ptr,    \
          max_num_partitions);


template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE,
          int NUM_THREADS = 128, int PARTITION_SIZE = 512>
void paged_attention_v2_launcher(
    torch::Tensor& out, torch::Tensor& exp_sums, torch::Tensor& max_logits,
    torch::Tensor& tmp_out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, float k_scale,
    float v_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  [[maybe_unused]] int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  float* exp_sums_ptr = reinterpret_cast<float*>(exp_sums.data_ptr());
  float* max_logits_ptr = reinterpret_cast<float*>(max_logits.data_ptr());
  T* tmp_out_ptr = reinterpret_cast<T*>(tmp_out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int max_num_partitions = DIVIDE_ROUND_UP(max_seq_len, PARTITION_SIZE);
  int logits_size = PARTITION_SIZE * sizeof(float);
  int V_VEC_SIZE = 16 / sizeof(CACHE_T);
  int NUM_V_VECS_PER_THREAD = head_size / V_VEC_SIZE;
  int NUM_COLS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_THREAD;
  int outputs_size = NUM_WARPS * head_size * sizeof(float) * NUM_COLS_PER_ITER;


  // For paged attention v2 kernel.
  dim3 grid(num_heads, num_seqs, max_num_partitions);
  int shared_mem_size = std::max(logits_size, outputs_size);
  // For paged attention v2 reduce kernel.
  dim3 reduce_grid(num_heads, num_seqs);
  int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);

  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model. However, we can easily extend this
    // to support any head size which is a multiple of 16.
    case 64:
      LAUNCH_PAGED_ATTENTION_V2_32N(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V2(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V2(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V2(112);
      break;
    case 120:
      LAUNCH_PAGED_ATTENTION_V2(120);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V2_32N(128);
      break;
    case 192:
      LAUNCH_PAGED_ATTENTION_V2(192);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V2_32N(256);
      break;
    default:
      TORCH_CHECK(false, "Unsupported head size: ", head_size);
      break;
  }
}

#define CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)   \
  paged_attention_v2_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,               \
                              IS_BLOCK_SPARSE>(                               \
      out, exp_sums, max_logits, tmp_out, query, key_cache, value_cache,      \
      num_kv_heads, scale, block_tables, seq_lens, max_seq_len, alibi_slopes, \
      k_scale, v_scale, tp_rank, blocksparse_local_blocks,                    \
      blocksparse_vert_stride, blocksparse_block_size,                        \
      blocksparse_head_sliding_step);

#define CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  switch (is_block_sparse) {                                               \
    case true:                                                             \
      CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);     \
      break;                                                               \
    case false:                                                            \
      CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);    \
      break;                                                               \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V2_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 8:                                                       \
      CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, 8, KV_DTYPE);         \
      break;                                                      \
    case 16:                                                      \
      CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    case 32:                                                      \
      CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, 32, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

void paged_attention_v2(
    torch::Tensor& out,         // [num_seqs, num_heads, head_size]
    torch::Tensor& exp_sums,    // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor& max_logits,  // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor&
        tmp_out,  // [num_seqs, num_heads, max_num_partitions, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double k_scale, double v_scale,
    const int64_t tp_rank, const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);
  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V2_LAUNCHER_BLOCK_SIZE)
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
